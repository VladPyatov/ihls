#include "fbmd_cuda.h"
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>
#include <hipsolver.h>

/**********************  Batch SymEig **************************************/

// Functions that compute the buffer size 

#define SYEVJ_BUFFER_ARGTYPES(T1, T2)                                        \
    hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo,\
    int m, const T1 *A, int lda, const T2 *E, int *lwork,                    \
    hipsolverSyevjInfo_t syevj_params, int batch_size


template <typename T1, typename T2>
hipsolverStatus_t _syevjBatched_bufferSize(
    SYEVJ_BUFFER_ARGTYPES(T1, T2));

template <>
hipsolverStatus_t _syevjBatched_bufferSize<float, float>(
    SYEVJ_BUFFER_ARGTYPES(float, float)){
    return hipsolverDnSsyevjBatched_bufferSize(
            handle, jobz, uplo, m, A, lda, E, lwork, syevj_params, batch_size);
}

template <>
hipsolverStatus_t _syevjBatched_bufferSize<double, double>(
    SYEVJ_BUFFER_ARGTYPES(double, double)){
    return hipsolverDnDsyevjBatched_bufferSize(
            handle, jobz, uplo, m, A, lda, E, lwork, syevj_params, batch_size);
    }

template <>
hipsolverStatus_t _syevjBatched_bufferSize<c10::complex<float>, float>(
    SYEVJ_BUFFER_ARGTYPES(c10::complex<float>, float)){
    return hipsolverDnCheevjBatched_bufferSize(
            handle, jobz, uplo, m, reinterpret_cast<const hipComplex*>(A),
            lda, E, lwork, syevj_params, batch_size);
    }
    
template <>
hipsolverStatus_t _syevjBatched_bufferSize<c10::complex<double>, double>(
    SYEVJ_BUFFER_ARGTYPES(c10::complex<double>, double)){
    return hipsolverDnZheevjBatched_bufferSize(
            handle, jobz, uplo, m, reinterpret_cast<const hipDoubleComplex*>(A),
            lda, E, lwork, syevj_params, batch_size);
    }

// Functions that compute the eigenvectors and eigenvalues

#define SYEVJ_ARGTYPES(T1, T2)                                               \
    hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo,\
    int m, T1 *A, int lda, T2 *E, T1 *work, int lwork, int *info,            \
    hipsolverSyevjInfo_t syevj_params, int batch_size

template <typename T1, typename T2>
hipsolverStatus_t _syevj_batched_solver(SYEVJ_ARGTYPES(T1, T2));

template <>
hipsolverStatus_t _syevj_batched_solver<float, float>(
    SYEVJ_ARGTYPES(float, float)) {
    return hipsolverDnSsyevjBatched(
            handle, jobz, uplo, m, A, lda, E, work, lwork, info, 
            syevj_params, batch_size);
    }

template <>
hipsolverStatus_t _syevj_batched_solver<double, double>(
    SYEVJ_ARGTYPES(double, double)) {
    return hipsolverDnDsyevjBatched(
            handle, jobz, uplo, m, A, lda, E, work, lwork, info, 
            syevj_params, batch_size);
    }

template <>
hipsolverStatus_t _syevj_batched_solver<c10::complex<float>, float>(
    SYEVJ_ARGTYPES(c10::complex<float>, float)) {
    return hipsolverDnCheevjBatched(
            handle, jobz, uplo, m, reinterpret_cast<hipComplex*>(A), lda, E,
            reinterpret_cast<hipComplex*>(work), lwork, info, syevj_params,
            batch_size);
    }
    
template <>
hipsolverStatus_t _syevj_batched_solver<c10::complex<double>, double>(
    SYEVJ_ARGTYPES(c10::complex<double>, double)) {
    return hipsolverDnZheevjBatched(
            handle, jobz, uplo, m, reinterpret_cast<hipDoubleComplex*>(A),
            lda, E, reinterpret_cast<hipDoubleComplex*>(work), lwork, info,
            syevj_params, batch_size);
    }

template <typename T1, typename T2>
void _batch_flattened_symeig_cuda_helper(
    T1 *d_A, T2 *d_E, int *info, const int batch_size, const int m,
    const int lda, const int sort, bool eigenvectors, bool upper,
    double tol=1e-7, int max_sweeps=100)
{
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverSyevjInfo_t syevj_params = NULL;
    
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat = hipSuccess;
    int lwork = 0; // size of workspace
    T1 *d_work = NULL; // device workspace for gesvdjBatched

    // Define whether to compute or not the eigenvectors
    const hipsolverEigMode_t jobz = 
        eigenvectors ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;
    // Define whether to use the upper or lower triangular part of the tensor
    const hipblasFillMode_t  uplo = 
        upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
    
    // Create cusolver handle, bind a stream  
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    // Configuration of syevj   
    status = hipsolverDnCreateSyevjInfo(&syevj_params);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
  
    status = hipsolverDnXsyevjSetTolerance(syevj_params, tol);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    status = hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    // Enable or disable sorting of the eigenvalues
    status = hipsolverDnXsyevjSetSortEig(syevj_params, sort);
    assert(HIPSOLVER_STATUS_SUCCESS == status);    
    
    // Query working space of syevjBatched 
    status = _syevjBatched_bufferSize<T1, T2>(
        cusolverH, jobz, uplo, m, d_A, lda, d_E, &lwork, syevj_params,
        batch_size);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    // Allocate work array
    cudaStat = hipMalloc((void**)&d_work, sizeof(T1)*lwork);
    assert(cudaStat == hipSuccess);

    // Compute eigenvectors and eigenvalues
    status = _syevj_batched_solver<T1, T2>(
        cusolverH, jobz, uplo, m, d_A, lda, d_E, d_work, lwork, info, 
        syevj_params, batch_size);
    cudaStat = hipDeviceSynchronize();            
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(cudaStat == hipSuccess);
    
    // free resources
    if (d_work) hipFree(d_work);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (stream) hipStreamDestroy(stream);
    if (syevj_params) hipsolverDnDestroySyevjInfo(syevj_params);    
}

std::tuple<at::Tensor, at::Tensor> _batch_flattened_symeig_cuda(
        at::Tensor &d_A, int sort, bool eigenvectors, bool upper, double tol, 
        int max_sweeps) {
    
    // Transform the tensor to a column-major layout so as to be compatible 
    // with CUBLAS.
    d_A = d_A.transpose(1,2).contiguous().transpose(1,2);
    
    const auto batch_size = d_A.size(0);
    const auto m = d_A.size(1);
    const auto lda = m;

    auto Dtype = (d_A.scalar_type() == torch::ScalarType::Float) ||
                 (d_A.scalar_type() == torch::ScalarType::ComplexFloat)
                 ? at::kFloat : at::kDouble;

    at::Tensor d_E = torch::empty({batch_size, m}, d_A.options().dtype(Dtype));
    at::Tensor info = torch::empty({batch_size}, d_A.options().dtype(at::kInt));

    switch (d_A.scalar_type()) {
        case torch::ScalarType::Double:
            _batch_flattened_symeig_cuda_helper<double>(
                d_A.data_ptr<double>(),
                d_E.data_ptr<double>(),
                info.data_ptr<int>(), batch_size, m, lda, sort, 
                eigenvectors, upper, tol, max_sweeps);
            break;

        case torch::ScalarType::Float:
            _batch_flattened_symeig_cuda_helper<float>(
                d_A.data_ptr<float>(),
                d_E.data_ptr<float>(),
                info.data_ptr<int>(), batch_size, m, lda, sort, 
                eigenvectors, upper, tol, max_sweeps);
            break;
        
        case torch::ScalarType::ComplexDouble:
            _batch_flattened_symeig_cuda_helper<c10::complex<double>, double>(
                d_A.data_ptr<c10::complex<double>>(),
                d_E.data_ptr<double>(),
                info.data_ptr<int>(), batch_size, m, lda, sort, 
                eigenvectors, upper, tol, max_sweeps);
            break;
            
        case torch::ScalarType::ComplexFloat:
            _batch_flattened_symeig_cuda_helper<c10::complex<float>, float>(
                d_A.data_ptr<c10::complex<float>>(),
                d_E.data_ptr<float>(),
                info.data_ptr<int>(), batch_size, m, lda, sort, 
                eigenvectors, upper, tol, max_sweeps);
            break;        

        default:
            AT_ERROR("This function doesn't support types other than "
                     "float, double, complexFloat, and complexDouble.");
    }

    // Check error status
    if (info.ne(0).any().item().toInt()){
        TORCH_WARN("CUSolver (syevj) did not converge");
    }

    return std::make_tuple(d_E, d_A);
}


/**********************  Batch SVD *****************************************/
    
// Functions that compute the buffer size 

#define GESVDJ_BUFFER_ARGTYPES(T1, T2)                                       \
    hipsolverHandle_t handle, hipsolverEigMode_t jobz, int m, int n,         \
    const T1 *A, int lda, const T2 *S, const T1 *U, int ldu, const T1 *V,    \
    int ldv, int *lwork, hipsolverGesvdjInfo_t gesvdj_params, int batch_size          \
    
template <typename T1, typename T2>
hipsolverStatus_t _gesvdjBatched_bufferSize(
    GESVDJ_BUFFER_ARGTYPES(T1, T2));

template <>
hipsolverStatus_t _gesvdjBatched_bufferSize<float, float>(
    GESVDJ_BUFFER_ARGTYPES(float, float)){
    return hipsolverDnSgesvdjBatched_bufferSize(
        handle, jobz, m, n, A, lda, S, U, ldu, V, ldv, lwork, 
        gesvdj_params, batch_size);
    }

template <>
hipsolverStatus_t _gesvdjBatched_bufferSize<double, double>(
    GESVDJ_BUFFER_ARGTYPES(double, double)){
    return hipsolverDnDgesvdjBatched_bufferSize(
        handle, jobz, m, n, A, lda, S, U, ldu, V, ldv, lwork, 
        gesvdj_params, batch_size);
    }

template <>
hipsolverStatus_t _gesvdjBatched_bufferSize<c10::complex<float>, float>(
    GESVDJ_BUFFER_ARGTYPES(c10::complex<float>, float)){
    return hipsolverDnCgesvdjBatched_bufferSize(
        handle, jobz, m, n, reinterpret_cast<const hipComplex*>(A), lda, S,
        reinterpret_cast<const hipComplex*>(U), ldu,
        reinterpret_cast<const hipComplex*>(V), ldv, lwork, gesvdj_params,
        batch_size);
    }

template <>
hipsolverStatus_t _gesvdjBatched_bufferSize<c10::complex<double>, double>(
    GESVDJ_BUFFER_ARGTYPES(c10::complex<double>, double)){
    return hipsolverDnZgesvdjBatched_bufferSize(
        handle, jobz, m, n, reinterpret_cast<const hipDoubleComplex*>(A), lda,
        S, reinterpret_cast<const hipDoubleComplex*>(U), ldu,
        reinterpret_cast<const hipDoubleComplex*>(V), ldv, lwork, gesvdj_params,
        batch_size);
    }

// Functions for computing the singular vectors and singular values

#define GESVDJ_ARGTYPES(T1, T2)                                              \
    hipsolverHandle_t handle, hipsolverEigMode_t jobz, int m, int n,         \
    T1 *A, int lda, T2 *S, T1 *U, int ldu, T1 *V, int ldv, T1 *work,         \
    int lwork, int *info, hipsolverGesvdjInfo_t gesvdj_params, int batch_size         \

template <typename T1, typename T2>
hipsolverStatus_t _gesvdj_batched_solver(GESVDJ_ARGTYPES(T1, T2));

template <>
hipsolverStatus_t _gesvdj_batched_solver<float, float>(
    GESVDJ_ARGTYPES(float, float)){
    return hipsolverDnSgesvdjBatched(
            handle, jobz, m, n, A, lda, S, U, ldu, V, ldv, work, lwork,
            info, gesvdj_params, batch_size);
    }

template <>
hipsolverStatus_t _gesvdj_batched_solver<double, double>(
    GESVDJ_ARGTYPES(double, double)){
    return hipsolverDnDgesvdjBatched(
            handle, jobz, m, n, A, lda, S, U, ldu, V, ldv, work, lwork,
            info, gesvdj_params, batch_size);
    }
    
template <>
hipsolverStatus_t _gesvdj_batched_solver<c10::complex<float>, float>(
    GESVDJ_ARGTYPES(c10::complex<float>, float)){
    return hipsolverDnCgesvdjBatched(
            handle, jobz, m, n, reinterpret_cast<hipComplex*>(A), lda, S,
            reinterpret_cast<hipComplex*>(U), ldu,
            reinterpret_cast<hipComplex*>(V), ldv,
            reinterpret_cast<hipComplex*>(work), lwork, info, gesvdj_params,
            batch_size);
    }

template <>
hipsolverStatus_t _gesvdj_batched_solver<c10::complex<double>, double>(
    GESVDJ_ARGTYPES(c10::complex<double>, double)){
    return hipsolverDnZgesvdjBatched(
            handle, jobz, m, n, reinterpret_cast<hipDoubleComplex*>(A), lda, S,
            reinterpret_cast<hipDoubleComplex*>(U), ldu,
            reinterpret_cast<hipDoubleComplex*>(V), ldv,
            reinterpret_cast<hipDoubleComplex*>(work), lwork, info,
            gesvdj_params, batch_size);
    }
    

template <typename T1, typename T2>
void _batch_flattened_svd_cuda_helper(
    T1 *d_A, T2 *d_S, T1 *d_U, T1 *d_V, int *info, const int batch_size,
    const int m, const int n, const int lda, const int ldu, const int ldv,
    const int sort_svd, bool compute_uv, double tol=1e-7, int max_sweeps=100)
{
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverGesvdjInfo_t gesvdj_params = NULL;
    
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat = hipSuccess;
    int lwork = 0; // size of workspace
    T1 *d_work = NULL; // device workspace for gesvdjBatched

    // Define whether to compute or not the singular vectors
    const hipsolverEigMode_t jobz = 
        compute_uv ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;

    // Create cusolver handle, bind a stream  
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    // Configuration of gesvdj   
    status = hipsolverDnCreateGesvdjInfo(&gesvdj_params);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
  
    status = hipsolverDnXgesvdjSetTolerance(gesvdj_params, tol);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    status = hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, max_sweeps);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    // Enable or disable sorting of the singular values
    status = hipsolverDnXgesvdjSetSortEig(gesvdj_params, sort_svd);
    assert(HIPSOLVER_STATUS_SUCCESS == status);    
    
    // Query working space of gesvdjBatched 
    status = _gesvdjBatched_bufferSize<T1, T2>(
            cusolverH, jobz, m, n, d_A, lda, d_S, d_U, ldu, d_V, ldv, 
            &lwork, gesvdj_params, batch_size);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    // Allocate work array
    cudaStat = hipMalloc((void**)&d_work, sizeof(T1)*lwork);
    assert(cudaStat == hipSuccess);

    // Compute singular vectors and singular values
    status = _gesvdj_batched_solver<T1, T2>(
            cusolverH, jobz, m, n, d_A, lda, d_S, d_U, ldu, d_V, ldv, 
            d_work, lwork, info, gesvdj_params, batch_size);
    cudaStat = hipDeviceSynchronize();            
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(cudaStat == hipSuccess);
    
    // free resources
    if (d_work) hipFree(d_work);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (stream) hipStreamDestroy(stream);
    if (gesvdj_params) hipsolverDnDestroyGesvdjInfo(gesvdj_params);    
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> _batch_flattened_svd_cuda(
        at::Tensor &A, int sort_svd, bool compute_uv, double tol,
        int max_sweeps) {
    
    // Transform tensor to a column-major layout so as to be compatible with CUBLAS.
    A = A.transpose(1,2).contiguous().transpose(1,2);
    
    const auto batch_size = A.size(0);
    const auto m = A.size(1);
    const auto n = A.size(2);
    const auto lda = m;
    const auto min_mn = (m < n)? m : n;
    const auto ldu = m;
    const auto ldv = n;

    auto Dtype = (A.scalar_type() == torch::ScalarType::Float) ||
                 (A.scalar_type() == torch::ScalarType::ComplexFloat)
                 ? at::kFloat : at::kDouble;

    at::Tensor d_U = torch::empty({batch_size, ldu, m}, A.options());
    at::Tensor d_S = torch::empty({batch_size, min_mn}, A.options().dtype(Dtype));
    at::Tensor d_V = torch::empty({batch_size, ldv, n}, A.options());
    at::Tensor info = torch::empty({batch_size}, A.options().dtype(at::kInt));

    switch (A.scalar_type()) {
        case torch::ScalarType::Double:
            _batch_flattened_svd_cuda_helper<double>(
                A.data_ptr<double>(),
                d_S.data_ptr<double>(),
                d_U.data_ptr<double>(),
                d_V.data_ptr<double>(),
                info.data_ptr<int>(), batch_size, m, n, lda, ldu, ldv, 
    		    sort_svd, compute_uv, tol, max_sweeps);
            break;

        case torch::ScalarType::Float:
            _batch_flattened_svd_cuda_helper<float>(
                A.data_ptr<float>(),
                d_S.data_ptr<float>(),
                d_U.data_ptr<float>(),
                d_V.data_ptr<float>(),
                info.data_ptr<int>(), batch_size, m, n, lda, ldu, ldv,
    		    sort_svd, compute_uv, tol, max_sweeps);
            break;
            
        case torch::ScalarType::ComplexDouble:
            _batch_flattened_svd_cuda_helper<c10::complex<double>, double>(
                A.data_ptr<c10::complex<double>>(),
                d_S.data_ptr<double>(),
                d_U.data_ptr<c10::complex<double>>(),
                d_V.data_ptr<c10::complex<double>>(),
                info.data_ptr<int>(), batch_size, m, n, lda, ldu, ldv, 
    		    sort_svd, compute_uv, tol, max_sweeps);
            break;

        case torch::ScalarType::ComplexFloat:
            _batch_flattened_svd_cuda_helper<c10::complex<float>, float>(
                A.data_ptr<c10::complex<float>>(),
                d_S.data_ptr<float>(),
                d_U.data_ptr<c10::complex<float>>(),
                d_V.data_ptr<c10::complex<float>>(),
                info.data_ptr<int>(), batch_size, m, n, lda, ldu, ldv, 
    		    sort_svd, compute_uv, tol, max_sweeps);
            break;

        default:
            AT_ERROR("This function doesn't support types other than "
                     "float, double, complexFloat, and complexDouble.");
    }

    // Check error status
    if (info.ne(0).any().item().toInt()){
        TORCH_WARN("CUSolver (gesvdj) did not converge");
    }

    return std::make_tuple(d_U, d_S, d_V);
}

